
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <sstream>

#define PI 3.14159265358979323846

__device__ double densityW(double Xold, double  Xnew, double sigma, double r, double delta, double delta_t){

double f=0, x=0;
//x=(1/(sigma*sqrt(delta_t)))*(log(Xnew)-log(Xold)-(r-delta-0.5*sigma*sigma)*delta_t);
x=(1/(sigma*sqrt(delta_t)))*(Xnew-Xold-(r-delta-0.5*sigma*sigma)*delta_t);
//f= (1/(sigma*sqrt(delta_t)*Xnew))*(1/(sqrt(2*PI)))*exp(-0.5*x*x); // this is the transition density
f= (1/(sigma*sqrt(delta_t)))*(1/(sqrt(2*PI)))*exp(-0.5*x*x);
return f;
}

__device__ double* two_dim_indexW(double* vector, int i, int j, double m, int b){

//int m_int= (int)m;
double* p;

//specify index layout here
p=&vector[b*(i)+(j)];

return p;


}

__device__ double* three_dim_indexW(double* matrix, int i, int j, int k, double m, int b, int num_assets){

//int m_int = (int)m;
double* p;

//specify index layout here
//p=&matrix[(m_int)*b*(k)+(m_int)*(j)+(i)];
p=&matrix[i*b*num_assets+j*num_assets+k];
return p;

}






__device__ double kahansum(double* sortvector, int b){
double sum=0, c=0, y, t;

        for(int i=0; i<b; i++){
                y=sortvector[i]-c;
                t=sum+y;
                c=(t-sum)-y;
                sum=t;
        }

return sum;
}


__global__ void valuesKernel(double* tempW_device ,double m,int  b, double* sigma_device,double* delta_device,double r, double delta_t,double* X_device,int num_assets){


int idx =blockDim.x*blockIdx.x + threadIdx.x;


int m_int=(int)m;
if(idx<(m_int-1)*b*b){

double w;

int i=idx/(b*b);
int j=idx/b;
if(j>(b-1)){
j=j%b;
}
int k=idx%b;


//              for(int k=0; k<b; k++){

//                      for(int j=0; j<b; j++){

                        //std::cout<<j<<std::endl;      
w=1;
                       //w=0; //set w to 1 since it will be equal to a product
for(int jjj=0; jjj<num_assets; jjj++){
	w = w * densityW(*three_dim_indexW(X_device, (i), k, jjj, m, b, num_assets), *three_dim_indexW(X_device, i+1, j, jjj, m, b, num_assets), sigma_device[jjj], r, delta_device[jjj], delta_t);
        }


tempW_device[idx]=w;

}
}


__global__ void sumweightsKernel(double* tempW_device , int b, double* weight_denominator_device, double m){

int idx =blockDim.x*blockIdx.x + threadIdx.x;
int m_int=(int)m;
if(idx<(m_int-1)*b){

double sum=0, c=0, y, t;

int start=idx*b;

for(int i=start; i<start+b; i++){
                y=tempW_device[i]-c;
                t=sum+y;
                c=(t-sum)-y;
                sum=t;
}

weight_denominator_device[idx]=sum;
}
}


__global__ void meshweightsKernel(double* W_device, double m, int b, double* sigma_device, double* delta_device, double r, double delta_t, double* X_device, int num_assets, double* weight_denominator_device, double* tempW_device){
double wdenominator;

int idx =blockDim.x*blockIdx.x + threadIdx.x;
int m_int=(int)m;

if(idx<b*b*m_int){

int i=idx/(b*b);
int k=idx/b;
if(k>(b-1)){
k=k%b;
}
int j=idx%b;


if(i==0){

                                if(j==0){

                                        *three_dim_indexW(W_device, i, k, j, m, b, b)=1;
                                }// all weights from the starting node are equal to 1

                                else{

                                        *three_dim_indexW(W_device, i, k, j, m, b, b)=0;
                                }
}	
		//dim1temp.clear();
if(i>0){		//sortvector.clear();


			//devide each element by the denominator
//			std::cout<<"before"<<std::endl;
			wdenominator= *two_dim_indexW(weight_denominator_device, i-1, k, m-1, b);
//			std::cout<<"after and I= "<<I<<std::endl;
//			std::cout<<*two_dim_index(weight_denominator, I, k, m-1, b)<<std::endl;
			//for(int t=0; t<b; t++){

			*three_dim_indexW(W_device, (i), k, j, m, b, b)=(((double)b) * (*three_dim_indexW(tempW_device, i-1, k, j, m-1, b, b)))/wdenominator;
		//	*three_dim_indexW(W_device, (i), k, j, m, b, b)=(((double)b)*(point))/wdenominator;
			//}
			//std::cout<<"after"<<std::endl;
		}

}
}

void meshweights(double* W, double m, int b, double sigma[], double delta[], double r, double delta_t, double* X, int num_assets, double* weight_denominator){
int m_int=(int)m;

int temp_N=(m_int-1) * b*b;

double* sigma_host;
sigma_host =sigma;
double* delta_host;
delta_host=delta;
double* tempW;
tempW= new double[temp_N];

//double* asset_amount_host;
//asset_amount_host =asset_amount;

int X_N=(m_int) * b * (num_assets);
int W_N=(m_int) * b*b;
int w_N=(m_int-1)*b;
int sigma_N=num_assets;
int delta_N=num_assets;

//int weight_denominator_N=(N-1) * b;

double* X_device;
double* W_device;
double* weight_denominator_device;
double* sigma_device;
double* delta_device;
double* tempW_device;

hipMalloc((void**) &X_device, X_N*sizeof(double) );
hipMemcpy(X_device, X, X_N*sizeof(double), hipMemcpyHostToDevice);

hipError_t error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &W_device, W_N*sizeof(double) );
hipMemcpy(W_device, W, W_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &sigma_device, sigma_N*sizeof(double) );
hipMemcpy(sigma_device, sigma_host, sigma_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &delta_device, delta_N*sizeof(double) );
hipMemcpy(delta_device, delta_host, delta_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &weight_denominator_device, w_N*sizeof(double) );
hipMemcpy(weight_denominator_device, weight_denominator, w_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &tempW_device, temp_N*sizeof(double) );
hipMemcpy(tempW_device, tempW, temp_N*sizeof(double), hipMemcpyHostToDevice);

//dim3 gridDim((int)ceil(temp_N/512.0));
//dim3 blockDim(512.0);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

hipDeviceSetLimit(hipLimitMallocHeapSize, 80000000*sizeof(double));

dim3 VgridDim((int)ceil(temp_N/512.0));
dim3 VblockDim(512.0);

valuesKernel<<<VgridDim,VblockDim>>>(tempW_device , m, b, sigma_device, delta_device, r, delta_t, X_device, num_assets);
hipDeviceSynchronize();

hipMemcpy(tempW, tempW_device, sizeof(double)*temp_N, hipMemcpyDeviceToHost);
hipMemcpy(tempW_device, tempW, temp_N*sizeof(double), hipMemcpyHostToDevice);

//dim3 gridDim((int)ceil(w_N/512.0));
//dim3 blockDim(512.0);
error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


//std::cout<<w_N/512<<std::endl;
dim3 sgridDim((int)ceil(w_N/512.0));
dim3 sblockDim(512.0);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


sumweightsKernel<<<sgridDim, sblockDim>>>(tempW_device , b, weight_denominator_device, m);

hipDeviceSynchronize();

hipMemcpy(weight_denominator, weight_denominator_device, sizeof(double)*w_N, hipMemcpyDeviceToHost);
hipMemcpy(weight_denominator_device, weight_denominator, w_N*sizeof(double), hipMemcpyHostToDevice);

//for(int check=0; check<w_N; check++){
//std::cout<< weight_denominator[check]<<std::endl;
//}

//dim3 gridDim((int)ceil(W_N/512.0));
//dim3 blockDim(512.0);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

dim3 mgridDim((int)ceil(W_N/512.0));
dim3 mblockDim(512.0);
meshweightsKernel<<<mgridDim, mblockDim>>>(W_device , m, b, sigma_device, delta_device, r, delta_t, X_device, num_assets, weight_denominator_device, tempW_device);

hipDeviceSynchronize();



 error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

hipMemcpy(W, W_device, sizeof(double)*W_N, hipMemcpyDeviceToHost);
hipMemcpy(weight_denominator, weight_denominator_device, sizeof(double)*w_N, hipMemcpyDeviceToHost);


hipFree(X_device);
hipFree(sigma_device);
hipFree(delta_device);
hipFree(W_device);
hipFree(weight_denominator_device);
hipFree(tempW_device);

delete[] tempW;
}


