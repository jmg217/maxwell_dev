#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include "enum_header.h" 
#include <unistd.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// REMEMBER TO PUT __host__ __device__ IN FRONT OF CLASS METHODS
  
#define PI 3.14159265358979323846


double* three_dim_index(double* matrix, int i, int j, int k, double m, int b, int num_assets);

__device__ double* two_dim_indexGPU(double* vector, int i, int j, double m, int b){

//int m_int= (int)m;
double* p;

//specify index layout here
p=&vector[b*(i)+(j)];

return p;


}

__device__ double* three_dim_indexGPU(double* matrix, int i, int j, int k, double m, int b, int num_assets){

int m_int = (int)m;
double* p;

//specify index layout here
//p=&matrix[(m_int)*b*(k)+(m_int)*(j)+(i)];
p=&matrix[i*b*num_assets+j*num_assets+k];
return p;

}

__device__ double densityGPU(double Xold, double  Xnew, double sigma, double r, double delta, double delta_t){

double f=0, x=0;
//x=(1/(sigma*sqrt(delta_t)))*(log(Xnew)-log(Xold)-(r-delta-0.5*sigma*sigma)*delta_t);
x=(1/(sigma*sqrt(delta_t)))*(Xnew-Xold-(r-delta-0.5*sigma*sigma)*delta_t);
//f= (1/(sigma*sqrt(delta_t)*Xnew))*(1/(sqrt(2*PI)))*exp(-0.5*x*x); // this is the transition density
f= (1/(sigma*sqrt(delta_t)))*(1/(sqrt(2*PI)))*exp(-0.5*x*x);
return f;

}
/*
__global__ void init(unsigned int seed, hiprandState_t* states) {
int idx=blockDim.x*blockIdx.x + threadIdx.x;

  // we have to initialize the state 
  hiprand_init(seed, // the seed can be the same for each core, here we pass the time in from the CPU 
              idx, // the sequence number should be different for each core (unless you want all
                            // cores to get the same sequence of numbers for some reason - use thread id! 
              0, // the offset is how much extra we advance in the sequence for each call, can be 0 
              &states[idx]);
}
*/

__device__ double GeometricPayOffCallV(double* X, double m, int b, int num_assets, double Strike){
double h;
h=1;
for(int l=0; l<num_assets; l++){
       // h*=exp(X[i][j][l]);

                //h*= exp(*two_dim_indexGPU(X, i, l, m, b));
		h*=exp(X[l]);
        }
        h=pow(h,1.0/(num_assets));
        if(h-Strike>0){
                h=h-Strike;
        }
        else{
                h=0;
        }
return h;
}

__device__ double GeometricPayOffPutV(double* X, double m, int b, int num_assets, double Strike){
double h;
h=1; 
for(int l=0; l<num_assets; l++){
       // h*=exp(X[i][j][l]);
                //h*= exp(*two_dim_indexGPU(X, i, l, m, b));
		h*=exp(X[l]);
        }
        h=pow(h,1.0/(num_assets));
        if(Strike-h>0){
                h=Strike-h;
        }
        else{
                h=0;
        }


return h;
}

__device__ void S_weights(double* S_Weights, double* X_device, double* S_new, int m, int b, double* sigma_device, double* delta_device, double delta_t, int num_assets, double r , int i, double* weight_denominator_device ){//note: S_new used to be just S
//if(blockDim.x*blockIdx.x + threadIdx.x==0){printf("Beginning \n");}

//double density_product,
double sum, w_s;

	for(int h=0; h<b; h++){   //h=k
	//if(blockDim.x*blockIdx.x + threadIdx.x==0){printf("Outside loop, i=%i \n", h);}


	sum=0;
	w_s=1;
		for(int kk=0; kk<num_assets; kk++){
			//w_s*=densityGPU(*two_dim_indexGPU(S, i, kk, m, num_assets), *three_dim_indexGPU(X_device, (i+1), h, kk, m, b), sigma_device[kk], r, delta_device[kk], delta_t);
			w_s*=densityGPU(S_new[kk], *three_dim_indexGPU(X_device, (i+1), h, kk, m, b, num_assets), sigma_device[kk], r, delta_device[kk], delta_t);

		}
/*
clock_t start_time =clock();

clock_t stop_time =clock();
int time=stop_time-start_time;
if(blockDim.x*blockIdx.x + threadIdx.x==0){printf("result at i=%i , = %i\n",i, time);}
*/
/*
	density_product=1;
	//if(blockDim.x*blockIdx.x + threadIdx.x==0){printf("after first inside loop \n");}

		for(int g=0; g<b; g++){   //g=l
//if(blockDim.x*blockIdx.x + threadIdx.x==0){printf("inside second loop i=%i \n", g);}

			for(int gg=0; gg<num_assets; gg++){

				density_product*=densityGPU(*three_dim_indexGPU(X_device, i, g, gg, m, b), *three_dim_indexGPU(X_device, (i+1), h, gg, m, b), sigma_device[gg], r, delta_device[gg], delta_t);

			}
		sum+=(1/((double)b))*density_product;
		}
*/
	sum = *two_dim_indexGPU(weight_denominator_device, i, h, m-1, b);
	if(sum==0){printf("division by zero in weights function of path estimator\n");}
	w_s = (((double)b)*w_s)/sum;	
//if(blockDim.x*blockIdx.x + threadIdx.x==0){printf("w_s=%f \n", w_s);}	
	//*two_dim_indexGPU(S_Weights, i, h, m, b)=w_s;
	S_Weights[h]=w_s;
	}

//if(blockDim.x*blockIdx.x + threadIdx.x==0){printf("End \n");}
}


__global__ void PathEstimatorKernel(double* X_device, double* weight_denominator_device, double* V_device, double* delta_device, double* sigma_device, double* X0_device, int N, double strike, double r, double delta_t, int b, int m, int num_assets, hiprandState_t* states, double* results_dev, double* asset_amount_device){


int idx =blockDim.x*blockIdx.x + threadIdx.x;
//if(blockDim.x*blockIdx.x + threadIdx.x==N+1){printf("n+1 outside \n");}
if(idx<N){
//printf("inside \n");
//if(blockDim.x*blockIdx.x + threadIdx.x==N-1){printf("n-1 \n");}
//if(blockDim.x*blockIdx.x + threadIdx.x==N+1){printf("n+1 inside \n");}

//GeometricPayOffPut thePayOff(strike);
//GeometricPayOffPut payoff(strike);

//enum Containers { vector, matrix };

//Containers Vector = vector;
//Containers Matrix = matrix;


double v_0, S_i, Z, C, H, sum, weight; //, w_s, sum_Z;
//srand((unsigned)time(NULL));
//std::random_device rd;
//std::default_random_engine generator;
//generator.seed( rd() );
//std::normal_distribution<double> distribution(0.0,1.0);
/// ARRAY CODE

//const int S_N=(m)*num_assets;
//const int S_W_N=(m)*b;

const int S_N= num_assets;
const int S_W_N= b; 

double* S_new;
S_new= new double[S_N]; 
//double s_new[S_new_N];
//S_new=s_new;

//double* S_old;
//S_old=new double[S_N];

double* S_Weights;
S_Weights=new double[S_W_N];
//double s_weights[S_W_new_N];
//S_Weights=s_weights;



//double* S_new;
//double* S_old;
//double* S_Weights;
/*
double s_new[1];
//double s_old[1];
double s_weights[250];
S_new=s_new;
//S_old=s_old;
S_Weights=s_weights;
*/
//S_Weights=new double[250];
//S_new=new double[1];



//if(idx==0){printf("X[0][0][0]= %f \n",*three_dim_indexGPU(X_device,0,0,0,m,b));}
//if(idx==0){printf("before the loop");}
int i=0;
do {

	if(i==0){
		for(int ll=0; ll<num_assets; ll++){
			//Z=boxmuller();
			// NEED TO CHANGE THE RANDOM NUMBER GENERATOR	
			//Z=distribution(generator);
			Z=hiprand_normal_double(&states[idx]);
//printf("for idx=%i, r=%f",idx,Z);
//printf("random number for idx %i is %f",idx,Z);

			S_i=X0_device[ll] +  (r-delta_device[ll]-0.5*pow(sigma_device[ll], 2))*delta_t + sigma_device[ll]*sqrt(delta_t)*Z;
			//tempnodevector.push_back(S_i);
			//*two_dim_indexGPU(S, i, ll, m, num_assets)=S_i;
			S_new[ll]=S_i;			
		}
	}

	else{
		for(int jj=0; jj<num_assets; jj++){
			//Z=boxmuller();
			//Z=distribution(generator);
			Z=hiprand_normal_double(&states[idx]);
			//if(idx==0){printf("random number=%f /n", Z);}
			//S_i=(*two_dim_indexGPU(S, (i-1), jj, m, num_assets)) +  (r-delta_device[jj]-0.5*pow(sigma_device[jj], 2))*delta_t + sigma_device[jj]*sqrt(delta_t)*Z;
			S_i=S_new[jj] + (r-delta_device[jj]-0.5*pow(sigma_device[jj], 2))*delta_t + sigma_device[jj]*sqrt(delta_t)*Z;
			//tempnodevector.push_back(S_i);
			//*two_dim_indexGPU(S, i, jj, m, num_assets)=S_i;
			S_new[jj]=S_i;
		}
	}
//printf("inside \n");
//if(idx==0){printf("before the call, m =%i /n", m);}
if(i<m-1){

//S_weights(tempvec, S_Weights, X, S, m, b, sigma, delta, delta_t, asset_amount, r, i  );
//S_weights(S_Weights, X_device, S, m, b, sigma_device, delta_device, delta_t, num_assets, r, i );
//right
S_weights(S_Weights, X_device, S_new, m, b, sigma_device, delta_device, delta_t, num_assets, r, i, weight_denominator_device);

}
//printf("inside \n");
double con_val=0; //continuation value variable
	sum=0;

	if(i==m-1){
	C=0;//continuation value at the last time step
	}
	
	else{
		for(int k=0; k<b; k++){	
			//weight= * two_dim_indexGPU(S_Weights, i, k, m, b);
			//right
			weight= S_Weights[k];
			
			//con_val=V[(m-1)-i-1][k];
			con_val= *two_dim_indexGPU(V_device, (m-1-i-1), k, m, b);
			//con_val=0;
			sum+=(weight) * (con_val); 			
		}
	
        //con_val=inner_product(b, first_vector, second_vector);
	
    
        C=(1/(double)b)*sum; //continuation value
//	C=(1/(double)b)*con_val;
	}	
	
//printf("inside \n");
//H=Payoff(S, strike, asset_amount, i)*exp(-r*delta_t*((i+1)));
//H=thePayOff(S, i, 0, m, num_assets, Vector, num_assets)*exp(-r*delta_t*((i+1)));
//H=0;
H= GeometricPayOffCallV(S_new, m, num_assets, num_assets, strike)*exp(-r*delta_t*((i+1)));

i=i+1;
/*for(int copy=0; copy<num_assets; copy++){
S_old[copy]=S_new[copy];
}*/
}while(H<C);//this will stop once H is less then the continuation value. at m-1, c=0 therefore m-1 is the max amount of loops.

v_0=H;

//if(idx==0){printf("result %i=%f", idx, v_0);}

results_dev[idx]=v_0;


delete[] S_new;
//delete[] S_old;
delete[] S_Weights;
//return v_0;
//printf("inside \n");
}



}

double PathEstimator(double strike, double r, double delta_t, int b, double m, double sigma[], double delta[], double X0[], double* X, double* weight_denominator, double* V, double asset_amount[], int num_assets, int Path_estimator_iterations, int iterator, int Final_iteration, hiprandState_t* States, hiprandState_t* states, int threads ){



//m=int(m);

//for(int test=0; test<((m-1)*b); test++){
//printf("at the start of pathestimator den=%f /n", weight_denominator[test]);
//}
//printf("Ib serial X[0][0][0]= %f \n",*three_dim_index(X,0,0,0,m,b));
hipError_t error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


int N= Path_estimator_iterations;

double* sigma_host;
sigma_host =sigma;

double* delta_host;
delta_host =delta;

double* X0_host;
X0_host =X0;

double* asset_amount_host;
asset_amount_host =asset_amount;

int m_int=(int)m;
//printf("at the start of pathestimator m_int=%i /n", m_int);


int X_N=(m_int) * b * (num_assets);
int W_N=(m_int-1) * b;
int V_N=(m_int) * b;
int delta_N= num_assets;
int sigma_N=num_assets;
int X0_N=num_assets;
int asset_amount_N = num_assets;

double* X_device;
double* V_device;
double* weight_denominator_device;
double* sigma_device;
double* delta_device;
double* X0_device;
double* asset_amount_device;



error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &X_device, X_N*sizeof(double) );
hipMemcpy(X_device, X, X_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &V_device, V_N*sizeof(double) );
hipMemcpy(V_device, V, V_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &weight_denominator_device, W_N*sizeof(double) );
hipMemcpy(weight_denominator_device, weight_denominator, W_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &X0_device, X0_N*sizeof(double) );
hipMemcpy(X0_device, X0_host, X0_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &sigma_device, sigma_N*sizeof(double) );
hipMemcpy(sigma_device, sigma_host, sigma_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &delta_device, delta_N*sizeof(double) );
hipMemcpy(delta_device, delta_host, delta_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &asset_amount_device, asset_amount_N*sizeof(double) );
hipMemcpy(asset_amount_device, asset_amount_host, asset_amount_N*sizeof(double), hipMemcpyHostToDevice);


hipMemcpy(states, States, threads*sizeof(hiprandState_t*), hipMemcpyHostToDevice);
//dim3 gridDim((int)ceil(N/512.0));
//printf("the grid dim is:%i\n",(int)ceil(N/512.0));
//dim3 blockDim(512);

dim3 gridDim((int)ceil(N/512.0));
dim3 blockDim(512.0);

/*if(N>512){
gridDim()= ceil(N/521);

}
*/
error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


double* results;
results = new double[N];
double* results_dev;
hipMalloc((void**) &results_dev, N*sizeof(double) );

// CALL RANDOM SEEDING KERNEL HERE
error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

/*
hiprandState_t* states;

hipMalloc((void**) &states, N * sizeof(hiprandState_t));

init<<<gridDim, blockDim>>>(time(0), states);

hipDeviceSynchronize();
*/
 
error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }
//printf("inside \n");
//hipDeviceSetLimit(hipLimitMallocHeapSize, 80000000*sizeof(double));
//size_t size;
//hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
    //printf("Heap size found to be %d\n",(int)size);
//printf("after");

//for(int test=0; test<V_N; test++){
//printf("N=%i, strike=%f, r=%f, delta_t=%f, num_a=%i, b=%i", N, strike, r, delta_t, num_assets,b);
//}
PathEstimatorKernel<<<gridDim, blockDim>>>(X_device, weight_denominator_device, V_device, delta_device, sigma_device, X0_device, N, strike, r, delta_t, b,  m_int, num_assets, states, results_dev, asset_amount_device);



hipDeviceSynchronize();


error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
	printf("found at line %d\n", __LINE__);
    exit(1);
  }
//printf("here");
hipMemcpy(results, results_dev, sizeof(double)*N, hipMemcpyDeviceToHost);

error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("Found at line %d\n", __LINE__);
    exit(1);
  }

//hipDeviceSynchronize();
//hipMemcpy(States, states, sizeof(hiprandState_t)*N, hipMemcpyDeviceToHost);
hipMemcpy(States, states, sizeof(hiprandState_t)*threads, hipMemcpyDeviceToHost);

error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("Found at line %d\n", __LINE__);
    exit(1);
  }


double result=0;
for(int f=0; f<Path_estimator_iterations; f++){
result+=results[f];
//printf("random %i =%f\n", f, results[f]);
}
result=(1/double(N))*result;

delete[] results;

error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipFree(X_device);
hipFree(V_device);
hipFree(weight_denominator_device);
hipFree(sigma_device);
hipFree(delta_device);
hipFree(X0_device);
hipFree(results_dev);
hipFree(asset_amount_device);

if(iterator==Final_iteration-1){
hipFree(states);
//printf("done, iter=%i",iterator);
}

//hipDeviceReset();
return result;
//hipDeviceReset();
}



