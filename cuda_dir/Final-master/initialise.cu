#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include "enum_header.h"
#include <unistd.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void init(unsigned int seed, hiprandState_t* states) {
int idx=blockDim.x*blockIdx.x + threadIdx.x;

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              idx, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[idx]);
}

void cuda_init_random(int N){

dim3 gridDim((int)ceil(N/512.0));
dim3 blockDim(512.0);

hiprandState_t* states;

hipMalloc((void**) &states, N * sizeof(hiprandState_t));

init<<<gridDim, blockDim>>>(time(0), states);

hipDeviceSynchronize();

hipError_t error = hipGetLastError();

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

}
