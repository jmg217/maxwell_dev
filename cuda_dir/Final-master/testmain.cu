#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>

 /* we need these includes for CUDA's random number stuff */
 #include <hiprand.h>
 #include <hiprand/hiprand_kernel.h>
#define N 5

 #define MAX 100

__global__ void init(unsigned int seed, hiprandState_t* states) {

   /* we have to initialize the state */
   hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
               threadIdx.x, /* the sequence number should be different for each core (unless you want all
 +                             cores to get the same sequence of numbers for some reason - use thread id! */
               0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
               &states[threadIdx.x]);
 }


void cuda_init_random(hiprandState_t* States, hiprandState_t* states){

//hiprandState_t* states;
//hiprandState_t* states;
//hipMalloc((void**) &states, N * sizeof(hiprandState_t));

   /* invoke the GPU to initialize all of the random states */
   init<<<1, N>>>(time(0), states);
hipDeviceSynchronize();
hipMemcpy(States, states, sizeof(hiprandState_t)*N, hipMemcpyDeviceToHost);

}

/*
 __global__ void randoms(hiprandState_t* states, double* numbers) {
   // hiprand works like rand - except that it takes a state as a parameter 
   numbers[threadIdx.x] = hiprand_normal_double(&states[threadIdx.x]);
 }
*/
/*
void cppfunc(int it, int final, hiprandState_t* States, hiprandState_t* states) {
//    CUDA's random number library uses hiprandState_t to keep track of the seed value
// +     we will store a random state for every thread  
//hiprandState_t* states;

//hiprandState_t* states;
//hipMalloc((void**) &states, N * sizeof(hiprandState_t));
   double cpu_nums[N];
   double* gpu_nums;
   hipMalloc((void**) &gpu_nums, N * sizeof(double));
// for(int it=0; it<10;it++){
printf("NEW BLOCK!!!!!!!!!!!!\n");
    invoke the kernel to get some random numbers 
   randoms<<<1, N>>>(states, gpu_nums);
hipDeviceSynchronize();
hipMemcpy(States, states, sizeof(hiprandState_t)*N, hipMemcpyDeviceToHost);
  //  copy the random numbers back 
   hipMemcpy(cpu_nums, gpu_nums, N * sizeof(double), hipMemcpyDeviceToHost);

   // print them out 
   for (int i = 0; i < N; i++) {
      printf("%lf\n", cpu_nums[i]);
   }
 //}
   // free the memory we allocated for the states and numbers 
hipFree(gpu_nums);

if(it==final){
   hipFree(states);
 } // hipFree(gpu_nums);

//return states;

 }
*/

void cppfunc(int it, int final, hiprandState_t* States, hiprandState_t* states);
//void cppfunc(int it, int final, hiprandState_t* states);

void hello_world(int it);

//void cuda_init_random(int N, hiprandState_t* states);

int main(){

hiprandState_t* States;
States= new hiprandState_t[N];
//hiprandState_t* states;
//hiprandState_t* states;
//hipMalloc((void**) &states, N * sizeof(hiprandState_t));
//cuda_init_random(States, states);
int it;
for( it=0; it<10; it++){

hiprandState_t* states;
//hiprandState_t* states;
hipMalloc((void**) &states, N * sizeof(hiprandState_t));
if(it==0){cuda_init_random(States, states);}
else{hipMemcpy(states, States, N*sizeof(hiprandState_t), hipMemcpyHostToDevice);}

cppfunc(it, 10, States, states);

hipDeviceReset();
}
hello_world(it);

delete[] States;
return 0;

}
