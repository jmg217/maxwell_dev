#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include "enum_header.h"
#include <unistd.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

double* three_dim_index(double* matrix, int i, int j, int k, double m, int b, int num_assets);

__device__ double* two_dim_indexGPUA(double* vector, int i, int j, double m, int b){

//int m_int= (int)m;
double* p;

//specify index layout here
p=&vector[b*(i)+(j)];

return p;


}

__device__ double* three_dim_indexGPUA(double* matrix, int i, int j, int k, double m, int b, int num_assets){

//int m_int = (int)m;
double* p;

//specify index layout here
//p=&matrix[(m_int)*b*(k)+(m_int)*(j)+(i)];
p=&matrix[i*b*num_assets+j*num_assets+k];
return p;

}


//__global__ void init(unsigned int seed, hiprandState_t* states);

__global__ void MeshGenKernel(double* X_device, double* delta_device, double* sigma_device,double* X0_device, int N, double strike, double r, double delta_t, int b, double m, int num_assets, hiprandState_t* states, double* asset_amount_device){

int idx =blockDim.x*blockIdx.x + threadIdx.x;

if(idx<N){
int m_int=(int)m;
/*int i,j,k;
i=floor(idx/(b*num_assets));
j=floor(idx/num_assets);
if(j>b-1){
j=j%b;
}
k=idx%num_assets;
*/
double Xi, Z;


for(int i=0; i<m_int; i++){
        if(i==0){
        
                for(int ll=0; ll<num_assets; ll++){
                       // Z=distribution(generator);
			Z=hiprand_normal_double(&states[idx]);
			//if(idx==0){printf("Z=%f\n",Z);}
        		*three_dim_indexGPUA(X_device, i, idx, ll, m, b, num_assets) = X0_device[ll] +  (r-delta_device[ll]-0.5*pow(sigma_device[ll], 2))*delta_t + sigma_device[ll]*sqrt(delta_t)*Z;

                }
        
        }

        if(i>0){
                for(int jj=0; jj<num_assets; jj++){
                        //Z=distribution(generator);
			Z=hiprand_normal_double(&states[idx]);
                        Xi=*three_dim_indexGPUA(X_device, (i-1), idx, jj, m, b, num_assets);
                        *three_dim_indexGPUA(X_device, i, idx, jj, m, b, num_assets)=Xi +  (r-delta_device[jj]-0.5*pow(sigma_device[jj], 2))*delta_t + sigma_device[jj]*sqrt(delta_t)*Z;
                }
        
        }
}

}
//else{hiprand_normal_double(&states[idx]);}
}
 
void mesh_generation(int b, int num_assets, double m, double X0[], double sigma[], double delta[], double asset_amount[], double* X, double strike, double r, double delta_t, hiprandState_t* States, hiprandState_t* states, int threads){

int m_int=(int)m;

int N= b;

double* X0_host;
X0_host =X0;

double* sigma_host;
sigma_host =sigma;

double* delta_host;
delta_host =delta;

double* asset_amount_host;
asset_amount_host =asset_amount;

int X_N=(m_int) * b * (num_assets);
int delta_N= num_assets;
int sigma_N=num_assets;
int X0_N=num_assets;
int asset_amount_N = num_assets;

double* X_device;
double* sigma_device;
double* delta_device;
double* X0_device;
double* asset_amount_device;

hipMalloc((void**) &X_device, X_N*sizeof(double) );
hipMemcpy(X_device, X, X_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &X0_device, X0_N*sizeof(double) );
hipMemcpy(X0_device, X0_host, X0_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &sigma_device, sigma_N*sizeof(double) );
hipMemcpy(sigma_device, sigma_host, sigma_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &delta_device, delta_N*sizeof(double) );
hipMemcpy(delta_device, delta_host, delta_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &asset_amount_device, asset_amount_N*sizeof(double) );
hipMemcpy(asset_amount_device, asset_amount_host, asset_amount_N*sizeof(double), hipMemcpyHostToDevice);


hipMemcpy(states, States, threads*sizeof(hiprandState_t*), hipMemcpyHostToDevice);

dim3 gridDim((int)ceil(N/512.0));
dim3 blockDim(512.0);

//hiprandState_t* states;

//hipMalloc((void**) &states, N * sizeof(hiprandState_t));

//init<<<gridDim, blockDim>>>(time(0), states);

//hipDeviceSynchronize();

MeshGenKernel<<<gridDim, blockDim>>>(X_device, delta_device, sigma_device, X0_device, N, strike, r, delta_t, b,  m, num_assets, states, asset_amount_device);

hipDeviceSynchronize();

hipMemcpy(X, X_device, sizeof(double)*X_N, hipMemcpyDeviceToHost);
hipError_t error = hipGetLastError(); 
 if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("Found at line %d\n", __LINE__);
    exit(1);
  }
hipMemcpy(States, states, sizeof(hiprandState_t)*threads, hipMemcpyDeviceToHost);

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("Found at line %d\n", __LINE__);
    exit(1);
  }


hipFree(X_device);
hipFree(sigma_device);
hipFree(delta_device);
hipFree(X0_device);
hipFree(asset_amount_device);

//hipDeviceReset();
}

