
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <sstream>

#define PI 3.14159265358979323846

//this function returns the transition densities between nodes
__device__ double densityW(double Xold, double  Xnew, double sigma, double r, double delta, double delta_t){

double f=0, x=0;

x=(1/(sigma*sqrt(delta_t)))*(Xnew-Xold-(r-delta-0.5*sigma*sigma)*delta_t);

f= (1/(sigma*sqrt(delta_t)))*(1/(sqrt(2*PI)))*exp(-0.5*x*x);
return f;
}

//this function provides a gpu index interface for 2-dim matrices stored as arrays
__device__ double* two_dim_indexW(double* vector, int i, int j, double m, int b){


double* p;

//specify index layout here
p=&vector[b*(i)+(j)];

return p;


}

//this function provides a gpu index interface for 3-dim matrices stored as arrays
__device__ double* three_dim_indexW(double* matrix, int i, int j, int k, double m, int b, int num_assets){


double* p;

//specify index layout here

p=&matrix[i*b*num_assets+j*num_assets+k];
return p;

}


//this kernel calculates the numerator values of the weight equation
__global__ void valuesKernel(double* tempW_device ,double m,int  b, double* sigma_device,double* delta_device,double r, double delta_t,double* X_device,int num_assets){


int idx =blockDim.x*blockIdx.x + threadIdx.x;


int m_int=(int)m;
if(idx<(m_int-1)*b*b){

double w;

int i=idx/(b*b);
int j=idx/b;
if(j>(b-1)){
j=j%b;
}
int k=idx%b;



w=1;

for(int jjj=0; jjj<num_assets; jjj++){
	w = w * densityW(*three_dim_indexW(X_device, (i), k, jjj, m, b, num_assets), *three_dim_indexW(X_device, i+1, j, jjj, m, b, num_assets), sigma_device[jjj], r, delta_device[jjj], delta_t);
        }


tempW_device[idx]=w;

}
}

//this kernel calculates the denominator values in the weight equation
__global__ void sumweightsKernel(double* tempW_device , int b, double* weight_denominator_device, double m){

int idx =blockDim.x*blockIdx.x + threadIdx.x;
int m_int=(int)m;
if(idx<(m_int-1)*b){

double sum=0, c=0, y, t;

int start=idx*b;

for(int i=start; i<start+b; i++){
                y=tempW_device[i]-c;
                t=sum+y;
                c=(t-sum)-y;
                sum=t;
}

weight_denominator_device[idx]=sum;
}
}

//this kernel calculates the mesh weights
__global__ void meshweightsKernel(double* W_device, double m, int b, double* sigma_device, double* delta_device, double r, double delta_t, double* X_device, int num_assets, double* weight_denominator_device, double* tempW_device){
double wdenominator;

int idx =blockDim.x*blockIdx.x + threadIdx.x;
int m_int=(int)m;

if(idx<b*b*m_int){

int i=idx/(b*b);
int k=idx/b;
if(k>(b-1)){
k=k%b;
}
int j=idx%b;


if(i==0){

                                if(j==0){

                                        *three_dim_indexW(W_device, i, k, j, m, b, b)=1;
                                }// all weights from the starting node are equal to 1

                                else{

                                        *three_dim_indexW(W_device, i, k, j, m, b, b)=0;
                                }
}	

if(i>0){		


			wdenominator= *two_dim_indexW(weight_denominator_device, i-1, k, m-1, b);
			*three_dim_indexW(W_device, (i), k, j, m, b, b)=(((double)b) * (*three_dim_indexW(tempW_device, i-1, k, j, m-1, b, b)))/wdenominator;

		}

}
}

//this function updates the weights matrix. it allocates memory on the device and initialises all the weights related kernels.
void meshweights(double* W, double m, int b, double sigma[], double delta[], double r, double delta_t, double* X, int num_assets, double* weight_denominator){
int m_int=(int)m;

int temp_N=(m_int-1) * b*b;

double* sigma_host;
sigma_host =sigma;
double* delta_host;
delta_host=delta;
double* tempW;
tempW= new double[temp_N];



int X_N=(m_int) * b * (num_assets);
int W_N=(m_int) * b*b;
int w_N=(m_int-1)*b;
int sigma_N=num_assets;
int delta_N=num_assets;

double* X_device;
double* W_device;
double* weight_denominator_device;
double* sigma_device;
double* delta_device;
double* tempW_device;

hipMalloc((void**) &X_device, X_N*sizeof(double) );
hipMemcpy(X_device, X, X_N*sizeof(double), hipMemcpyHostToDevice);

hipError_t error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &W_device, W_N*sizeof(double) );
hipMemcpy(W_device, W, W_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &sigma_device, sigma_N*sizeof(double) );
hipMemcpy(sigma_device, sigma_host, sigma_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &delta_device, delta_N*sizeof(double) );
hipMemcpy(delta_device, delta_host, delta_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &weight_denominator_device, w_N*sizeof(double) );
hipMemcpy(weight_denominator_device, weight_denominator, w_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


hipMalloc((void**) &tempW_device, temp_N*sizeof(double) );
hipMemcpy(tempW_device, tempW, temp_N*sizeof(double), hipMemcpyHostToDevice);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

hipDeviceSetLimit(hipLimitMallocHeapSize, 80000000*sizeof(double));

dim3 VgridDim((int)ceil(temp_N/512.0));
dim3 VblockDim(512.0);

valuesKernel<<<VgridDim,VblockDim>>>(tempW_device , m, b, sigma_device, delta_device, r, delta_t, X_device, num_assets);
hipDeviceSynchronize();

hipMemcpy(tempW, tempW_device, sizeof(double)*temp_N, hipMemcpyDeviceToHost);
hipMemcpy(tempW_device, tempW, temp_N*sizeof(double), hipMemcpyHostToDevice);


error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }



dim3 sgridDim((int)ceil(w_N/512.0));
dim3 sblockDim(512.0);

error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }


sumweightsKernel<<<sgridDim, sblockDim>>>(tempW_device , b, weight_denominator_device, m);

hipDeviceSynchronize();

hipMemcpy(weight_denominator, weight_denominator_device, sizeof(double)*w_N, hipMemcpyDeviceToHost);
hipMemcpy(weight_denominator_device, weight_denominator, w_N*sizeof(double), hipMemcpyHostToDevice);



error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

dim3 mgridDim((int)ceil(W_N/512.0));
dim3 mblockDim(512.0);
meshweightsKernel<<<mgridDim, mblockDim>>>(W_device , m, b, sigma_device, delta_device, r, delta_t, X_device, num_assets, weight_denominator_device, tempW_device);

hipDeviceSynchronize();



 error = hipGetLastError();


  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("found at line %d\n", __LINE__);
    exit(1);
  }

hipMemcpy(W, W_device, sizeof(double)*W_N, hipMemcpyDeviceToHost);
hipMemcpy(weight_denominator, weight_denominator_device, sizeof(double)*w_N, hipMemcpyDeviceToHost);


hipFree(X_device);
hipFree(sigma_device);
hipFree(delta_device);
hipFree(W_device);
hipFree(weight_denominator_device);
hipFree(tempW_device);

delete[] tempW;
}


