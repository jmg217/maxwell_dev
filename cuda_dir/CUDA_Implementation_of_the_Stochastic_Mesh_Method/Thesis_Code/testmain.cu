#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>

 /* we need these includes for CUDA's random number stuff */
 #include <hiprand.h>
 #include <hiprand/hiprand_kernel.h>
#define N 5

 #define MAX 100

__global__ void init(unsigned int seed, hiprandState_t* states) {

   /* we have to initialize the state */
   hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
               threadIdx.x, /* the sequence number should be different for each core (unless you want all
 +                             cores to get the same sequence of numbers for some reason - use thread id! */
               0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
               &states[threadIdx.x]);
 }


void cuda_init_random(hiprandState_t* States, hiprandState_t* states){

//hiprandState_t* states;
//hiprandState_t* states;
//hipMalloc((void**) &states, N * sizeof(hiprandState_t));

   /* invoke the GPU to initialize all of the random states */
   init<<<1, N>>>(time(0), states);
hipDeviceSynchronize();
hipMemcpy(States, states, sizeof(hiprandState_t)*N, hipMemcpyDeviceToHost);

}

void cppfunc(int it, int final, hiprandState_t* States, hiprandState_t* states);
//void cppfunc(int it, int final, hiprandState_t* states);

void hello_world(int it);

//void cuda_init_random(int N, hiprandState_t* states);

int main(){

hiprandState_t* States;
States= new hiprandState_t[N];
//hiprandState_t* states;
//hiprandState_t* states;
//hipMalloc((void**) &states, N * sizeof(hiprandState_t));
//cuda_init_random(States, states);
int it;
for( it=0; it<10; it++){

hiprandState_t* states;
//hiprandState_t* states;
hipMalloc((void**) &states, N * sizeof(hiprandState_t));
if(it==0){cuda_init_random(States, states);}
else{hipMemcpy(states, States, N*sizeof(hiprandState_t), hipMemcpyHostToDevice);}

cppfunc(it, 10, States, states);

hipDeviceReset();
}
hello_world(it);

delete[] States;
return 0;

}
