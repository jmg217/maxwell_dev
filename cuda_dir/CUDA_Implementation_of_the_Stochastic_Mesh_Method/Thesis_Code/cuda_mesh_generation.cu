#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include "enum_header.h"
#include <unistd.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


__host__ __device__ double* three_dim_index(double* matrix, int i, int j, int k, double m, int b, int num_assets);


__host__ __device__ double* two_dim_index(double* vector, int i, int j, double m, int b);


//this kernel generates the stochastic mesh.
__global__ void MeshGenKernel(double* X_device, double* delta_device, double* sigma_device,double* X0_device, int N, double strike, double r, double delta_t, int b, double m, int num_assets, hiprandState_t* states, double* asset_amount_device){
//thread index
int idx =blockDim.x*blockIdx.x + threadIdx.x;

if(idx<N){
int m_int=(int)m;
double Xi, Z;

//we set the values of each mesh node in this loop
for(int i=0; i<m_int; i++){
        if(i==0){
        
                for(int ll=0; ll<num_assets; ll++){
                       
			Z=hiprand_normal_double(&states[idx]);
			
        		*three_dim_index(X_device, i, idx, ll, m, b, num_assets) = X0_device[ll] +  (r-delta_device[ll]-0.5*pow(sigma_device[ll], 2))*delta_t + sigma_device[ll]*sqrt(delta_t)*Z;

                }
        
        }

        if(i>0){
                for(int jj=0; jj<num_assets; jj++){
                        
			Z=hiprand_normal_double(&states[idx]);
                        Xi=*three_dim_index(X_device, (i-1), idx, jj, m, b, num_assets);
                        *three_dim_index(X_device, i, idx, jj, m, b, num_assets)=Xi +  (r-delta_device[jj]-0.5*pow(sigma_device[jj], 2))*delta_t + sigma_device[jj]*sqrt(delta_t)*Z;
                }
        
        }
}

}

}
 
void mesh_generation(int b, int num_assets, double m, double X0[], double sigma[], double delta[], double asset_amount[], double* X, double strike, double r, double delta_t, hiprandState_t* States, hiprandState_t* states, int threads){

int m_int=(int)m;

int N= b;

double* X0_host;
X0_host =X0;

double* sigma_host;
sigma_host =sigma;

double* delta_host;
delta_host =delta;

double* asset_amount_host;
asset_amount_host =asset_amount;

int X_N=(m_int) * b * (num_assets);
int delta_N= num_assets;
int sigma_N=num_assets;
int X0_N=num_assets;
int asset_amount_N = num_assets;

double* X_device;
double* sigma_device;
double* delta_device;
double* X0_device;
double* asset_amount_device;

//allocate memory on the device and copy from cpu to device

hipMalloc((void**) &X_device, X_N*sizeof(double) );
hipMemcpy(X_device, X, X_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &X0_device, X0_N*sizeof(double) );
hipMemcpy(X0_device, X0_host, X0_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &sigma_device, sigma_N*sizeof(double) );
hipMemcpy(sigma_device, sigma_host, sigma_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &delta_device, delta_N*sizeof(double) );
hipMemcpy(delta_device, delta_host, delta_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &asset_amount_device, asset_amount_N*sizeof(double) );
hipMemcpy(asset_amount_device, asset_amount_host, asset_amount_N*sizeof(double), hipMemcpyHostToDevice);


hipMemcpy(states, States, threads*sizeof(hiprandState_t*), hipMemcpyHostToDevice);

//set the number of threads
dim3 gridDim((int)ceil(N/512.0));
dim3 blockDim(512.0);

//launch kernel
MeshGenKernel<<<gridDim, blockDim>>>(X_device, delta_device, sigma_device, X0_device, N, strike, r, delta_t, b,  m, num_assets, states, asset_amount_device);

hipDeviceSynchronize();

hipMemcpy(X, X_device, sizeof(double)*X_N, hipMemcpyDeviceToHost);
hipError_t error = hipGetLastError(); 
 if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("Found at line %d\n", __LINE__);
    exit(1);
  }
hipMemcpy(States, states, sizeof(hiprandState_t)*threads, hipMemcpyDeviceToHost);

  if( error != hipSuccess )
  {
    std::cout << hipGetErrorString(error) << std::endl;
    printf("Found at line %d\n", __LINE__);
    exit(1);
  }

//free the gpu memory

hipFree(X_device);
hipFree(sigma_device);
hipFree(delta_device);
hipFree(X0_device);
hipFree(asset_amount_device);


}

