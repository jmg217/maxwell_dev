
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <sstream>

#define PI 3.14159265358979323846

__device__ double density(double Xold, double  Xnew, double sigma, double r, double delta, double delta_t){

double f=0, x=0;
//x=(1/(sigma*sqrt(delta_t)))*(log(Xnew)-log(Xold)-(r-delta-0.5*sigma*sigma)*delta_t);
x=(1/(sigma*sqrt(delta_t)))*(Xnew-Xold-(r-delta-0.5*sigma*sigma)*delta_t);
//f= (1/(sigma*sqrt(delta_t)*Xnew))*(1/(sqrt(2*PI)))*exp(-0.5*x*x); // this is the transition density
f= (1/(sigma*sqrt(delta_t)))*(1/(sqrt(2*PI)))*exp(-0.5*x*x);
return f;
}

