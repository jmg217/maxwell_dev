#include "hip/hip_runtime.h"
/* This code will multiply two vectors and
   check the result.
*/

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <iostream>
#include <stdio.h>

#define CUDA_CHECK {hipDeviceSynchronize();	\
  hipError_t err = hipGetLastError();\
  if(err){\
    std::cout << "Error: " << hipGetErrorString(err) << " line " << __LINE__ << std::endl; \
    exit(1);\
  }}

//#define CUDA_CHECK

/* Fill in your dotProduct kernel here...
 */

// Naive (and wrong) way
__global__ void calcDotProductKern1(float *x, float *y, float *res, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < N){
    (*res) += x[i] * y[i];
  }
}

// Naive, correct, but slow way
__global__ void calcDotProductKern2(float *x, float *y, float *res, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < N){
    atomicAdd(res, x[i] * y[i]);
  }
}

// Better reduction first pass
// res needs to point to at least 'blocks' floats.
__global__ void calcDotProductKern3(float *x, float *y, float *res, int N)
{
  __shared__ float product[512];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int locI = threadIdx.x;
  if(i < N)
    {
      product[locI] = x[i] * y[i];
    }
  else{
    product[locI] = 0;
  }
  __syncthreads();

  int blockSize = blockDim.x;

  if( blockSize >= 1024 && locI < 512){
    product[locI] = product[locI] + product[locI+512];
    __syncthreads();
  }
  if( blockSize >= 512 && locI < 256){
    product[locI] = product[locI] + product[locI+256];
    __syncthreads();
  }
  if( blockSize >= 256 && locI < 128){
    product[locI] = product[locI] + product[locI+128];
    __syncthreads();
  }
  if( blockSize >= 128 && locI < 64){
    product[locI] = product[locI] + product[locI+64];
    __syncthreads();
  }
  if( blockSize >= 64 && locI < 32){
    product[locI] = product[locI] + product[locI+32];
    __syncthreads();
  }
  if( blockSize >= 32 && locI < 16){
    product[locI] = product[locI] + product[locI+16];
    __syncthreads();
  }
  if( blockSize >= 16 && locI < 8){
    product[locI] = product[locI] + product[locI+8];
  }
  if( blockSize >= 8 && locI < 4){
    product[locI] = product[locI] + product[locI+4];
  }
  if( blockSize >= 4 && locI < 2){
    product[locI] = product[locI] + product[locI+2];
  }
  if( blockSize >= 2 && locI < 1){
    product[locI] = product[locI] + product[locI+1];
  }
  if( locI == 0){
    res[blockIdx.x] = product[0];
  }
}

// Generic reduction
// x[] is of size N, and y[] is of size N/blockDim.x
__global__ void reduce(float *x, float *y, int N)
{
  __shared__ float result[512];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int locI = threadIdx.x;
  int blockSize = blockDim.x;

  if(i < N){
    result[locI] = x[i];
  }else{
    result[locI] = 0;
  }
  __syncthreads();
  if( blockSize >= 1024 && locI < 512){
    result[locI] = result[locI] + result[locI+512];
    __syncthreads();
  }
  if( blockSize >= 512 && locI < 256){
    result[locI] = result[locI] + result[locI+256];
    __syncthreads();
  }
  if( blockSize >= 256 && locI < 128){
    result[locI] = result[locI] + result[locI+128];
    __syncthreads();
  }
  if( blockSize >= 128 && locI < 64){
    result[locI] = result[locI] + result[locI+64];
    __syncthreads();
  }
  if( blockSize >= 64 && locI < 32){
    result[locI] = result[locI] + result[locI+32];
    __syncthreads();
  }
  if( blockSize >= 32 && locI < 16){
    result[locI] = result[locI] + result[locI+16];
    __syncthreads();
  }
  if( blockSize >= 16 && locI < 8){
    result[locI] = result[locI] + result[locI+8];
  }
  if( blockSize >= 8 && locI < 4){
    result[locI] = result[locI] + result[locI+4];
  }
  if( blockSize >= 4 && locI < 2){
    result[locI] = result[locI] + result[locI+2];
  }
  if( blockSize >= 2 && locI < 1){
    result[locI] = result[locI] + result[locI+1];
  }
  if(locI == 0){
    y[blockIdx.x] = result[0];
  }
}

float calcDotProduct1(float* x, float* y, int N){

  int threads = 512;
  int blocks = (N + threads - 1)/ threads;

  float* res;
  hipMalloc(&res, sizeof(float));
  float resHost = 0;
  hipMemcpy(res, &resHost, sizeof(float), hipMemcpyHostToDevice);
  CUDA_CHECK;
  calcDotProductKern1<<<blocks, threads>>>(x, y, res, N);
  CUDA_CHECK;
  hipMemcpy(&resHost, res, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(res);
  CUDA_CHECK;
  return resHost;
}

float calcDotProduct2(float* x, float* y, int N){

  int threads = 512;
  int blocks = (N + threads - 1)/ threads;

  float* res;
  hipMalloc(&res, sizeof(float));
  float resHost = 0;
  hipMemcpy(res, &resHost, sizeof(float), hipMemcpyHostToDevice);
  CUDA_CHECK;
  calcDotProductKern2<<<blocks, threads>>>(x, y, res, N);
  CUDA_CHECK;
  hipMemcpy(&resHost, res, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(res);
  CUDA_CHECK;
  return resHost;
}

float calcDotProduct3(float* x, float* y, int N){
  int threads = 512;
  int blocks = (N + threads - 1)/ threads;

  float* res;
  hipMalloc(&res, blocks*sizeof(float));
  CUDA_CHECK;
  calcDotProductKern3<<<blocks, threads>>>(x, y, res, N);
  CUDA_CHECK;
  float* resHost = new float[blocks];
  hipMemcpy(resHost, res, sizeof(float) * blocks, hipMemcpyDeviceToHost);
  CUDA_CHECK;
  float p=0;
  for(int i=0 ; i < blocks ; i++){
    p += resHost[i];
  }

  delete[] resHost;
  hipFree(res);
  CUDA_CHECK;
  return p;
}


float calcDotProduct3Reduce(float* x, float* y, int N){
  int threads = 512;
  int blocks = (N + threads - 1)/ threads;

  float* res;
  hipMalloc(&res, blocks*sizeof(float));
  CUDA_CHECK;
  calcDotProductKern3<<<blocks, threads>>>(x, y, res, N);
  CUDA_CHECK;
  while(blocks > 1){
    int blocksOrig = blocks;
    blocks = ceil((float)blocks / threads);
    float* resOrig = res;
    hipMalloc(&res, blocks*sizeof(float));
    reduce<<<blocks, threads>>>(resOrig, res, blocksOrig);

    hipFree(resOrig);
  }
  CUDA_CHECK;
  float resHost;
  hipMemcpy(&resHost, res, sizeof(float), hipMemcpyDeviceToHost);
  CUDA_CHECK;
  hipFree(res);
  return resHost;
}

float calcDotProductThrust(float* x, float* y, int N){
  thrust::device_ptr<float> xThStart(x);
  thrust::device_ptr<float> yThStart(y);
  thrust::device_ptr<float> xThEnd(x + N);
  thrust::device_ptr<float> yThEnd(y + N);

  return thrust::inner_product(xThStart, xThEnd, yThStart, 0.0f);
}

float timeDotProduct(float (*kernel)(float*, float*, int), float *x, float *y, int N, float ans)
{
  CUDA_CHECK;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  CUDA_CHECK;
  hipEventRecord(start, 0);
  
  double prod = kernel(x, y, N);

  hipEventRecord(end, 0);
  CUDA_CHECK;
  hipEventSynchronize(end);
  CUDA_CHECK;
  hipError_t err = hipGetLastError();

  if(err){
    std::cout << "Error: " << hipGetErrorString(err) << std::endl;
  }

  if( fabs(prod - ans) / fabs(ans) < 1e-4 )
  {
    std::cout << "Multiplication correct! " << prod << " = " << ans << std::endl;
  }
  else
  {
    std::cout << "Multiplication wrong! " << prod << " != " << ans << std::endl;
  }

  float timeInMs;
  hipEventElapsedTime(&timeInMs, start, end);
  std::cout << "Time: " << timeInMs << "ms" << std::endl << std::endl;

  CUDA_CHECK;

  hipEventDestroy(start);
  hipEventDestroy(end);

  CUDA_CHECK;
  
  return 0;
}


int main(void)
{
  const int N = 20000000;

  float *x_host = new float[N];
  float *y_host = new float[N];
  

  // Fill matrix and vector on host
  for(int i=0 ; i < N ; i++)
  {
    x_host[i] = sin(i*0.013);
    y_host[i] = cos(i*0.019);
  }
  
  float *x;
  float *y;

  hipMalloc(&x, N*sizeof(float));
  hipMalloc(&y, N*sizeof(float));

  CUDA_CHECK;
  
  // Copy x and y to device
  hipMemcpy(x, x_host, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y, y_host, N*sizeof(float), hipMemcpyHostToDevice);  

  CUDA_CHECK;

  //hipMemcpy(y_host, y, N*sizeof(float), hipMemcpyDeviceToHost);

  // Check result

  clock_t st = clock();
  float prod = 0;
  for(int i=0 ; i < N ; i++)
  {
    prod += y_host[i] * x_host[i];
  }
  clock_t end = clock();

  std::cout << "CPU time = " << (end - st) / (float)CLOCKS_PER_SEC * 1000 << " ms" << std::endl;

  std::cout << "Naive approach - wrong" << std::endl;
  timeDotProduct(calcDotProduct1, x, y, N, prod);
  std::cout << "Using atomic operations" << std::endl;
  timeDotProduct(calcDotProduct2, x, y, N, prod);
  std::cout << "Reduction across one thread block only" << std::endl;
  timeDotProduct(calcDotProduct3, x, y, N, prod);
  std::cout << "Repeated reduction" << std::endl;
  timeDotProduct(calcDotProduct3Reduce, x, y, N, prod);
  std::cout << "Thrust" << std::endl;
  timeDotProduct(calcDotProductThrust, x, y, N, prod);

  hipFree(x);
  hipFree(y);

  delete[] x_host;
  delete[] y_host;
}
